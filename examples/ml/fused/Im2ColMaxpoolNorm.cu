#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/NativeFunctions.h>
#include <ATen/TensorUtils.h>
#include <ATen/Utils.h>
#include <ATen/div_rtn.h>

#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>

#include "Im2ColMaxpoolNorm.cuh"

namespace at {
namespace native {

std::tuple<Tensor, Tensor, Tensor, Tensor, Tensor, Tensor> im2col_maxpool_batch_norm_stream(
    const Tensor& input,
    IntArrayRef kernel_size,
    IntArrayRef dilation,
    IntArrayRef padding,
    IntArrayRef stride,
    const Tensor& input_maxpool_,
    IntArrayRef kernel_size_maxpool,
    IntArrayRef stride_maxpool,
    IntArrayRef padding_maxpool,
    IntArrayRef dilation_maxpool,
    bool ceil_mode,
    const Tensor& input_batch_norm) {
  // auto r2 = AT_DISPATCH_FLOATING_TYPES_AND_HALF(input_batch_norm.scalar_type(), "batch_norm_stats_cuda", [&] {
  //   return im2col_batch_norm_fused<scalar_t, int32_t>(
  //     input, kernel_size, dilation, padding, stride,
  //     input_batch_norm, 0.1);
  // });
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(input_batch_norm.scalar_type(), "batch_norm_stats_cuda", [&] {
    return im2col_maxpool_batch_norm_stream<scalar_t, int32_t>(
      input, kernel_size, dilation, padding, stride,
      input_maxpool_,
      kernel_size_maxpool,
      stride_maxpool,
      padding_maxpool,
      dilation_maxpool,
      ceil_mode,
      input_batch_norm, 0.1);
  });
  auto r2 = AT_DISPATCH_FLOATING_TYPES_AND_HALF(input_batch_norm.scalar_type(), "batch_norm_stats_cuda", [&] {
    return im2col_maxpool_batch_norm_fused<scalar_t, int32_t>(
      input, kernel_size, dilation, padding, stride,
      input_maxpool_,
      kernel_size_maxpool,
      stride_maxpool,
      padding_maxpool,
      dilation_maxpool,
      ceil_mode,
      input_batch_norm, 0.1);
  });
  return std::tuple_cat(r2, r2);
}


} // namespace native
} // namespace at
