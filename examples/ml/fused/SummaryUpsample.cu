#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>

#include <THC/THCGeneral.h>
#include <THC/THCDeviceUtils.cuh>

#include <ATen/ATen.h>
#include <ATen/TensorUtils.h>
#include <ATen/Utils.h>

#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/detail/KernelUtils.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/detail/TensorInfo.cuh>
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/NativeFunctions.h>
#include <ATen/TensorUtils.h>
#include <ATen/Utils.h>
#include <ATen/div_rtn.h>

#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>

#include <c10/macros/Macros.h>
#include <ATen/native/im2col_shape_check.h>
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/NativeFunctions.h>
#include <ATen/TensorUtils.h>
#include <ATen/Utils.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include "../cuda/UpSample.cuh"
#include "../cuda/DeviceSqrt.cuh"
#include "../cuda/LaunchUtils.h"

#include <hip/hip_runtime_api.h>
namespace at {
namespace native {


__device__ __forceinline__ size_t
idx(const size_t nc,
    const size_t height,
    const size_t width,
    const size_t y,
    const size_t x) {
  return (nc * height + y) * width + x;
}

template <typename scalar_t, typename accscalar_t>
__global__ void upsample_bilinear2d_out_frame(
    const int n,
    const accscalar_t rheight,
    const accscalar_t rwidth,
    const bool align_corners,
    const PackedTensorAccessor<scalar_t, 4> idata,
    PackedTensorAccessor<scalar_t, 4> odata) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;

  const int batchsize = idata.size(0);
  const int channels = idata.size(1);
  const int height1 = idata.size(2);
  const int width1 = idata.size(3);
  const int height2 = odata.size(2);
  const int width2 = odata.size(3);

  if (index < n) {
    const int w2 = index % width2; // 0:width2-1
    const int h2 = index / width2; // 0:height2-1
    // special case: just copy
    if (height1 == height2 && width1 == width2) {
      const int h1 = h2;
      const int w1 = w2;
      for (int n = 0; n < batchsize; n++) {
        for (int c = 0; c < channels; ++c) {
          const scalar_t val = idata[n][c][h1][w1];
          odata[n][c][h2][w2] = val;
        }
      }
      return;
    }
    //
    const accscalar_t h1r = area_pixel_compute_source_index<accscalar_t>(
        rheight, h2, align_corners, /*cubic=*/false);
    const int h1 = h1r;
    const int h1p = (h1 < height1 - 1) ? 1 : 0;
    const accscalar_t h1lambda = h1r - h1;
    const accscalar_t h0lambda = static_cast<accscalar_t>(1) - h1lambda;
    //
    const accscalar_t w1r = area_pixel_compute_source_index<accscalar_t>(
        rwidth, w2, align_corners, /*cubic=*/false);
    const int w1 = w1r;
    const int w1p = (w1 < width1 - 1) ? 1 : 0;
    const accscalar_t w1lambda = w1r - w1;
    const accscalar_t w0lambda = static_cast<accscalar_t>(1) - w1lambda;
    //
    for (int n = 0; n < batchsize; n++) {
      for (int c = 0; c < channels; ++c) {
        const accscalar_t val = h0lambda *
                (w0lambda * idata[n][c][h1][w1] +
                 w1lambda * idata[n][c][h1][w1 + w1p]) +
            h1lambda *
                (w0lambda * idata[n][c][h1 + h1p][w1] +
                 w1lambda * idata[n][c][h1 + h1p][w1 + w1p]);
        odata[n][c][h2][w2] = static_cast<scalar_t>(val);
      }
    }
  }
}


using namespace at::cuda;
using namespace at::cuda::detail;

#define THRESH_NUMBER_BINS_FOR_MULTI_BLOCK_MEM 100
#define THRESH_NUMBER_BINS_FOR_GLOBAL_MEM 1000
#define FOR_KERNEL_LOOP(i, lim)                                      \
  for (IndexType i = blockIdx.x * blockDim.x + threadIdx.x; i < lim; \
       i += gridDim.x * blockDim.x)

/*
  Memory types used for the 3 histogram implementations.
  See `CUDA_tensor_histogram` below.
 */
enum class CUDAHistogramMemoryType { SHARED, MULTI_BLOCK, GLOBAL };
namespace {
  template<typename input_t, typename IndexType>
  __device__ static IndexType getBin(input_t bVal, input_t minvalue, input_t maxvalue, int nbins) {
    IndexType bin = (int)((bVal - minvalue) * nbins / (maxvalue - minvalue));
    // (only applicable for histc)
    // while each bin is inclusive at the lower end and exclusive at the higher, i.e. [start, end)
    // the last bin is inclusive at both, i.e. [start, end], in order to include maxvalue if exists
    // therefore when bin == nbins, adjust bin to the last bin
    if (bin == nbins) bin -= 1;
    return bin;
  }
}

template <typename output_t31, typename input_t32, typename IndexType33, int ADims34, int PDims35, int BDims36, at::native::CUDAHistogramMemoryType MemoryType37 = CUDAHistogramMemoryType::MULTI_BLOCK, typename Op38, typename scalar_t0, typename accscalar_t1>
void kernelHistogram1D_upsample_bilinear2d_out_frame_0(TensorInfo<output_t31, IndexType33> a39, TensorInfo<output_t31, IndexType33> p40, TensorInfo<input_t32, IndexType33> b41, int nbins42, input_t32 minvalue43, input_t32 maxvalue44, IndexType33 totalElements45, Op38 getOp46, const int n2, const accscalar_t1 rheight3, const accscalar_t1 rwidth4, const bool align_corners5, const PackedTensorAccessor<scalar_t0, 4> idata6, PackedTensorAccessor<scalar_t0, 4> odata7) __attribute__((global))
 {
if (!((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y)>=0 && (threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) < 512)) goto label_0;
unsigned int blockDim_x_1;
blockDim_x_1 = 512;
unsigned int threadIdx_x_1;
threadIdx_x_1 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 0) % 512;
unsigned int blockDim_y_1;
blockDim_y_1 = 1;
unsigned int threadIdx_y_1;
threadIdx_y_1 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 0) / 512 % 1;
unsigned int blockDim_z_1;
blockDim_z_1 = 1;
unsigned int threadIdx_z_1;
threadIdx_z_1 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 0) / 512;
extern unsigned char my_smem47[] __attribute__((shared));
output_t31 *smem48;
smem48 = nullptr;
smem48 = reinterpret_cast<output_t31 *>(my_smem47);
for (IndexType33 i = threadIdx_x_1; i < a39.sizes[0]; i += blockDim_x_1) {
    smem48[i] = 0;
}
label_0:;
__syncthreads();
if (!((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y)>=0 && (threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) < 512)) goto label_1;
for (IndexType33 linearIndex = blockIdx.x * blockDim_x_1 + threadIdx_x_1; linearIndex < totalElements45; linearIndex += gridDim.x * blockDim_x_1) {
    IndexType33 bOffset49;
    bOffset49 = IndexToOffset<input_t32, IndexType33, BDims36>::get(linearIndex, b41);
    input_t32 bVal50;
    bVal50 = b41.data[bOffset49];
    if (bVal50 >= minvalue43 && bVal50 <= maxvalue44) {
        IndexType33 bin51;
        bin51 = getBin<input_t32, IndexType33>(bVal50, minvalue43, maxvalue44, nbins42);
        atomicAdd(&smem48[bin51], getOp46(linearIndex));
    }
}
label_1:;
__syncthreads();
if (!((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y)>=0 && (threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) < 512)) goto label_2;
for (IndexType33 i = threadIdx_x_1; i < a39.sizes[0]; i += blockDim_x_1) {
    IndexType33 aOffset52;
    aOffset52 = IndexToOffset<output_t31, IndexType33, ADims34>::get(i, a39);
    atomicAdd(&a39.data[aOffset52], smem48[i]);
}
label_2:;
if (!((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y)>=512 && (threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) < 1024)) goto label_3;
unsigned int blockDim_x_0;
blockDim_x_0 = 512;
unsigned int threadIdx_x_0;
threadIdx_x_0 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 512) % 512;
unsigned int blockDim_y_0;
blockDim_y_0 = 1;
unsigned int threadIdx_y_0;
threadIdx_y_0 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 512) / 512 % 1;
unsigned int blockDim_z_0;
blockDim_z_0 = 1;
unsigned int threadIdx_z_0;
threadIdx_z_0 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 512) / 512;
int index8;
index8 = threadIdx_x_0 + blockIdx.x * blockDim_x_0;
int batchsize9;
batchsize9 = idata6.size(0);
int channels10;
channels10 = idata6.size(1);
int height111;
height111 = idata6.size(2);
int width112;
width112 = idata6.size(3);
int height213;
height213 = odata7.size(2);
int width214;
width214 = odata7.size(3);
if (index8 < n2) {
    int w215;
    w215 = index8 % width214;
    int h216;
    h216 = index8 / width214;
    if (height111 == height213 && width112 == width214) {
        int h127;
        h127 = h216;
        int w128;
        w128 = w215;
        for (int n = 0; n < batchsize9; n++) {
            for (int c = 0; c < channels10; ++c) {
                scalar_t0 val29;
                val29 = idata6[n][c][h127][w128];
                odata7[n][c][h216][w215] = val29;
            }
        }
        return;
    }
    accscalar_t1 h1r17;
    h1r17 = area_pixel_compute_source_index<accscalar_t1>(rheight3, h216, align_corners5, false);
    int h118;
    h118 = h1r17;
    int h1p19;
    h1p19 = (h118 < height111 - 1) ? 1 : 0;
    accscalar_t1 h1lambda20;
    h1lambda20 = h1r17 - h118;
    accscalar_t1 h0lambda21;
    h0lambda21 = static_cast<accscalar_t1>(1) - h1lambda20;
    accscalar_t1 w1r22;
    w1r22 = area_pixel_compute_source_index<accscalar_t1>(rwidth4, w215, align_corners5, false);
    int w123;
    w123 = w1r22;
    int w1p24;
    w1p24 = (w123 < width112 - 1) ? 1 : 0;
    accscalar_t1 w1lambda25;
    w1lambda25 = w1r22 - w123;
    accscalar_t1 w0lambda26;
    w0lambda26 = static_cast<accscalar_t1>(1) - w1lambda25;
    for (int n = 0; n < batchsize9; n++) {
        for (int c = 0; c < channels10; ++c) {
            accscalar_t1 val30;
            val30 = h0lambda21 * (w0lambda26 * idata6[n][c][h118][w123] + w1lambda25 * idata6[n][c][h118][w123 + w1p24]) + h1lambda20 * (w0lambda26 * idata6[n][c][h118 + h1p19][w123] + w1lambda25 * idata6[n][c][h118 + h1p19][w123 + w1p24]);
            odata7[n][c][h216][w215] = static_cast<scalar_t0>(val30);
        }
    }
}
label_3:;
}
template <typename output_t31, typename input_t32, typename IndexType33, int ADims34, int PDims35, int BDims36, at::native::CUDAHistogramMemoryType MemoryType37 = CUDAHistogramMemoryType::MULTI_BLOCK, typename Op38, typename scalar_t0, typename accscalar_t1>
void kernelHistogram1D_upsample_bilinear2d_out_frame_11(TensorInfo<output_t31, IndexType33> a39, TensorInfo<output_t31, IndexType33> p40, TensorInfo<input_t32, IndexType33> b41, int nbins42, input_t32 minvalue43, input_t32 maxvalue44, IndexType33 totalElements45, Op38 getOp46, const int n2, const accscalar_t1 rheight3, const accscalar_t1 rwidth4, const bool align_corners5, const PackedTensorAccessor<scalar_t0, 4> idata6, PackedTensorAccessor<scalar_t0, 4> odata7) __attribute__((global))
 {
if (!((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y)>=0 && (threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) < 512)) goto label_0;
unsigned int blockDim_x_1;
blockDim_x_1 = 512;
unsigned int threadIdx_x_1;
threadIdx_x_1 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 0) % 512;
unsigned int blockDim_y_1;
blockDim_y_1 = 1;
unsigned int threadIdx_y_1;
threadIdx_y_1 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 0) / 512 % 1;
unsigned int blockDim_z_1;
blockDim_z_1 = 1;
unsigned int threadIdx_z_1;
threadIdx_z_1 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 0) / 512;
extern unsigned char my_smem47[] __attribute__((shared));
output_t31 *smem48;
smem48 = nullptr;
smem48 = reinterpret_cast<output_t31 *>(my_smem47);
for (IndexType33 i = threadIdx_x_1; i < a39.sizes[0]; i += blockDim_x_1) {
    smem48[i] = 0;
}
label_0:;
__syncthreads();
if (!((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y)>=0 && (threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) < 512)) goto label_1;
for (IndexType33 linearIndex = blockIdx.x * blockDim_x_1 + threadIdx_x_1; linearIndex < totalElements45; linearIndex += gridDim.x * blockDim_x_1) {
    IndexType33 bOffset49;
    bOffset49 = IndexToOffset<input_t32, IndexType33, BDims36>::get(linearIndex, b41);
    input_t32 bVal50;
    bVal50 = b41.data[bOffset49];
    if (bVal50 >= minvalue43 && bVal50 <= maxvalue44) {
        IndexType33 bin51;
        bin51 = getBin<input_t32, IndexType33>(bVal50, minvalue43, maxvalue44, nbins42);
        atomicAdd(&smem48[bin51], getOp46(linearIndex));
    }
}
label_1:;
if (!((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y)>=512 && (threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) < 1024)) goto label_3;
unsigned int blockDim_x_0;
blockDim_x_0 = 512;
unsigned int threadIdx_x_0;
threadIdx_x_0 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 512) % 512;
unsigned int blockDim_y_0;
blockDim_y_0 = 1;
unsigned int threadIdx_y_0;
threadIdx_y_0 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 512) / 512 % 1;
unsigned int blockDim_z_0;
blockDim_z_0 = 1;
unsigned int threadIdx_z_0;
threadIdx_z_0 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 512) / 512;
int index8;
index8 = threadIdx_x_0 + blockIdx.x * blockDim_x_0;
int batchsize9;
batchsize9 = idata6.size(0);
int channels10;
channels10 = idata6.size(1);
int height111;
height111 = idata6.size(2);
int width112;
width112 = idata6.size(3);
int height213;
height213 = odata7.size(2);
int width214;
width214 = odata7.size(3);
if (index8 < n2) {
    int w215;
    w215 = index8 % width214;
    int h216;
    h216 = index8 / width214;
    if (height111 == height213 && width112 == width214) {
        int h127;
        h127 = h216;
        int w128;
        w128 = w215;
        for (int n = 0; n < batchsize9; n++) {
            for (int c = 0; c < channels10; ++c) {
                scalar_t0 val29;
                val29 = idata6[n][c][h127][w128];
                odata7[n][c][h216][w215] = val29;
            }
        }
        return;
    }
    accscalar_t1 h1r17;
    h1r17 = area_pixel_compute_source_index<accscalar_t1>(rheight3, h216, align_corners5, false);
    int h118;
    h118 = h1r17;
    int h1p19;
    h1p19 = (h118 < height111 - 1) ? 1 : 0;
    accscalar_t1 h1lambda20;
    h1lambda20 = h1r17 - h118;
    accscalar_t1 h0lambda21;
    h0lambda21 = static_cast<accscalar_t1>(1) - h1lambda20;
    accscalar_t1 w1r22;
    w1r22 = area_pixel_compute_source_index<accscalar_t1>(rwidth4, w215, align_corners5, false);
    int w123;
    w123 = w1r22;
    int w1p24;
    w1p24 = (w123 < width112 - 1) ? 1 : 0;
    accscalar_t1 w1lambda25;
    w1lambda25 = w1r22 - w123;
    accscalar_t1 w0lambda26;
    w0lambda26 = static_cast<accscalar_t1>(1) - w1lambda25;
    for (int n = 0; n < batchsize9; n++) {
        for (int c = 0; c < channels10; ++c) {
            accscalar_t1 val30;
            val30 = h0lambda21 * (w0lambda26 * idata6[n][c][h118][w123] + w1lambda25 * idata6[n][c][h118][w123 + w1p24]) + h1lambda20 * (w0lambda26 * idata6[n][c][h118 + h1p19][w123] + w1lambda25 * idata6[n][c][h118 + h1p19][w123 + w1p24]);
            odata7[n][c][h216][w215] = static_cast<scalar_t0>(val30);
        }
    }
}
label_3:;
__syncthreads();
if (!((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y)>=0 && (threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) < 512)) goto label_2;
for (IndexType33 i = threadIdx_x_1; i < a39.sizes[0]; i += blockDim_x_1) {
    IndexType33 aOffset52;
    aOffset52 = IndexToOffset<output_t31, IndexType33, ADims34>::get(i, a39);
    atomicAdd(&a39.data[aOffset52], smem48[i]);
}
label_2:;
}
template <typename output_t31, typename input_t32, typename IndexType33, int ADims34, int PDims35, int BDims36, at::native::CUDAHistogramMemoryType MemoryType37 = CUDAHistogramMemoryType::MULTI_BLOCK, typename Op38, typename scalar_t0, typename accscalar_t1>
void kernelHistogram1D_upsample_bilinear2d_out_frame_100(TensorInfo<output_t31, IndexType33> a39, TensorInfo<output_t31, IndexType33> p40, TensorInfo<input_t32, IndexType33> b41, int nbins42, input_t32 minvalue43, input_t32 maxvalue44, IndexType33 totalElements45, Op38 getOp46, const int n2, const accscalar_t1 rheight3, const accscalar_t1 rwidth4, const bool align_corners5, const PackedTensorAccessor<scalar_t0, 4> idata6, PackedTensorAccessor<scalar_t0, 4> odata7) __attribute__((global))
 {
if (((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y)>=0 && (threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) < 512)){
    unsigned int blockDim_x_1;
    blockDim_x_1 = 512;
    unsigned int threadIdx_x_1;
    threadIdx_x_1 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 0) % 512;
    unsigned int blockDim_y_1;
    blockDim_y_1 = 1;
    unsigned int threadIdx_y_1;
    threadIdx_y_1 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 0) / 512 % 1;
    unsigned int blockDim_z_1;
    blockDim_z_1 = 1;
    unsigned int threadIdx_z_1;
    threadIdx_z_1 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 0) / 512;
    extern unsigned char my_smem47[] __attribute__((shared));
    output_t31 *smem48;
    smem48 = nullptr;
    smem48 = reinterpret_cast<output_t31 *>(my_smem47);
    for (IndexType33 i = threadIdx_x_1; i < a39.sizes[0]; i += blockDim_x_1) {
        smem48[i] = 0;
    }
    __syncthreads();
    for (IndexType33 linearIndex = blockIdx.x * blockDim_x_1 + threadIdx_x_1; linearIndex < totalElements45; linearIndex += gridDim.x * blockDim_x_1) {
        IndexType33 bOffset49;
        bOffset49 = IndexToOffset<input_t32, IndexType33, BDims36>::get(linearIndex, b41);
        input_t32 bVal50;
        bVal50 = b41.data[bOffset49];
        if (bVal50 >= minvalue43 && bVal50 <= maxvalue44) {
            IndexType33 bin51;
            bin51 = getBin<input_t32, IndexType33>(bVal50, minvalue43, maxvalue44, nbins42);
            atomicAdd(&smem48[bin51], getOp46(linearIndex));
        }
    }
    __syncthreads();
    for (IndexType33 i = threadIdx_x_1; i < a39.sizes[0]; i += blockDim_x_1) {
        IndexType33 aOffset52;
        aOffset52 = IndexToOffset<output_t31, IndexType33, ADims34>::get(i, a39);
        atomicAdd(&a39.data[aOffset52], smem48[i]);
    }
}
if (((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y)>=0 && (threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) < 512)){
    unsigned int blockDim_x_0;
    blockDim_x_0 = 512;
    unsigned int threadIdx_x_0;
    threadIdx_x_0 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 0) % 512;
    unsigned int blockDim_y_0;
    blockDim_y_0 = 1;
    unsigned int threadIdx_y_0;
    threadIdx_y_0 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 0) / 512 % 1;
    unsigned int blockDim_z_0;
    blockDim_z_0 = 1;
    unsigned int threadIdx_z_0;
    threadIdx_z_0 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 0) / 512;
    int index8;
    index8 = threadIdx_x_0 + blockIdx.x * blockDim_x_0;
    int batchsize9;
    batchsize9 = idata6.size(0);
    int channels10;
    channels10 = idata6.size(1);
    int height111;
    height111 = idata6.size(2);
    int width112;
    width112 = idata6.size(3);
    int height213;
    height213 = odata7.size(2);
    int width214;
    width214 = odata7.size(3);
    if (index8 < n2) {
        int w215;
        w215 = index8 % width214;
        int h216;
        h216 = index8 / width214;
        if (height111 == height213 && width112 == width214) {
            int h127;
            h127 = h216;
            int w128;
            w128 = w215;
            for (int n = 0; n < batchsize9; n++) {
                for (int c = 0; c < channels10; ++c) {
                    scalar_t0 val29;
                    val29 = idata6[n][c][h127][w128];
                    odata7[n][c][h216][w215] = val29;
                }
            }
            return;
        }
        accscalar_t1 h1r17;
        h1r17 = area_pixel_compute_source_index<accscalar_t1>(rheight3, h216, align_corners5, false);
        int h118;
        h118 = h1r17;
        int h1p19;
        h1p19 = (h118 < height111 - 1) ? 1 : 0;
        accscalar_t1 h1lambda20;
        h1lambda20 = h1r17 - h118;
        accscalar_t1 h0lambda21;
        h0lambda21 = static_cast<accscalar_t1>(1) - h1lambda20;
        accscalar_t1 w1r22;
        w1r22 = area_pixel_compute_source_index<accscalar_t1>(rwidth4, w215, align_corners5, false);
        int w123;
        w123 = w1r22;
        int w1p24;
        w1p24 = (w123 < width112 - 1) ? 1 : 0;
        accscalar_t1 w1lambda25;
        w1lambda25 = w1r22 - w123;
        accscalar_t1 w0lambda26;
        w0lambda26 = static_cast<accscalar_t1>(1) - w1lambda25;
        for (int n = 0; n < batchsize9; n++) {
            for (int c = 0; c < channels10; ++c) {
                accscalar_t1 val30;
                val30 = h0lambda21 * (w0lambda26 * idata6[n][c][h118][w123] + w1lambda25 * idata6[n][c][h118][w123 + w1p24]) + h1lambda20 * (w0lambda26 * idata6[n][c][h118 + h1p19][w123] + w1lambda25 * idata6[n][c][h118 + h1p19][w123 + w1p24]);
                odata7[n][c][h216][w215] = static_cast<scalar_t0>(val30);
            }
        }
    }
}
}
template <typename output_t31, typename input_t32, typename IndexType33, int ADims34, int PDims35, int BDims36, at::native::CUDAHistogramMemoryType MemoryType37 = CUDAHistogramMemoryType::MULTI_BLOCK, typename Op38, typename scalar_t0, typename accscalar_t1>
void kernelHistogram1D_upsample_bilinear2d_out_frame_2(TensorInfo<output_t31, IndexType33> a39, TensorInfo<output_t31, IndexType33> p40, TensorInfo<input_t32, IndexType33> b41, int nbins42, input_t32 minvalue43, input_t32 maxvalue44, IndexType33 totalElements45, Op38 getOp46, const int n2, const accscalar_t1 rheight3, const accscalar_t1 rwidth4, const bool align_corners5, const PackedTensorAccessor<scalar_t0, 4> idata6, PackedTensorAccessor<scalar_t0, 4> odata7) __attribute__((global))
 {
if (!((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y)>=0 && (threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) < 512)) goto label_0;
unsigned int blockDim_x_1;
blockDim_x_1 = 512;
unsigned int threadIdx_x_1;
threadIdx_x_1 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 0) % 512;
unsigned int blockDim_y_1;
blockDim_y_1 = 1;
unsigned int threadIdx_y_1;
threadIdx_y_1 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 0) / 512 % 1;
unsigned int blockDim_z_1;
blockDim_z_1 = 1;
unsigned int threadIdx_z_1;
threadIdx_z_1 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 0) / 512;
extern unsigned char my_smem47[] __attribute__((shared));
output_t31 *smem48;
smem48 = nullptr;
smem48 = reinterpret_cast<output_t31 *>(my_smem47);
for (IndexType33 i = threadIdx_x_1; i < a39.sizes[0]; i += blockDim_x_1) {
    smem48[i] = 0;
}
label_0:;
if (!((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y)>=512 && (threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) < 1024)) goto label_3;
unsigned int blockDim_x_0;
blockDim_x_0 = 512;
unsigned int threadIdx_x_0;
threadIdx_x_0 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 512) % 512;
unsigned int blockDim_y_0;
blockDim_y_0 = 1;
unsigned int threadIdx_y_0;
threadIdx_y_0 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 512) / 512 % 1;
unsigned int blockDim_z_0;
blockDim_z_0 = 1;
unsigned int threadIdx_z_0;
threadIdx_z_0 = ((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) - 512) / 512;
int index8;
index8 = threadIdx_x_0 + blockIdx.x * blockDim_x_0;
int batchsize9;
batchsize9 = idata6.size(0);
int channels10;
channels10 = idata6.size(1);
int height111;
height111 = idata6.size(2);
int width112;
width112 = idata6.size(3);
int height213;
height213 = odata7.size(2);
int width214;
width214 = odata7.size(3);
if (index8 < n2) {
    int w215;
    w215 = index8 % width214;
    int h216;
    h216 = index8 / width214;
    if (height111 == height213 && width112 == width214) {
        int h127;
        h127 = h216;
        int w128;
        w128 = w215;
        for (int n = 0; n < batchsize9; n++) {
            for (int c = 0; c < channels10; ++c) {
                scalar_t0 val29;
                val29 = idata6[n][c][h127][w128];
                odata7[n][c][h216][w215] = val29;
            }
        }
        return;
    }
    accscalar_t1 h1r17;
    h1r17 = area_pixel_compute_source_index<accscalar_t1>(rheight3, h216, align_corners5, false);
    int h118;
    h118 = h1r17;
    int h1p19;
    h1p19 = (h118 < height111 - 1) ? 1 : 0;
    accscalar_t1 h1lambda20;
    h1lambda20 = h1r17 - h118;
    accscalar_t1 h0lambda21;
    h0lambda21 = static_cast<accscalar_t1>(1) - h1lambda20;
    accscalar_t1 w1r22;
    w1r22 = area_pixel_compute_source_index<accscalar_t1>(rwidth4, w215, align_corners5, false);
    int w123;
    w123 = w1r22;
    int w1p24;
    w1p24 = (w123 < width112 - 1) ? 1 : 0;
    accscalar_t1 w1lambda25;
    w1lambda25 = w1r22 - w123;
    accscalar_t1 w0lambda26;
    w0lambda26 = static_cast<accscalar_t1>(1) - w1lambda25;
    for (int n = 0; n < batchsize9; n++) {
        for (int c = 0; c < channels10; ++c) {
            accscalar_t1 val30;
            val30 = h0lambda21 * (w0lambda26 * idata6[n][c][h118][w123] + w1lambda25 * idata6[n][c][h118][w123 + w1p24]) + h1lambda20 * (w0lambda26 * idata6[n][c][h118 + h1p19][w123] + w1lambda25 * idata6[n][c][h118 + h1p19][w123 + w1p24]);
            odata7[n][c][h216][w215] = static_cast<scalar_t0>(val30);
        }
    }
}
label_3:;
__syncthreads();
if (!((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y)>=0 && (threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) < 512)) goto label_1;
for (IndexType33 linearIndex = blockIdx.x * blockDim_x_1 + threadIdx_x_1; linearIndex < totalElements45; linearIndex += gridDim.x * blockDim_x_1) {
    IndexType33 bOffset49;
    bOffset49 = IndexToOffset<input_t32, IndexType33, BDims36>::get(linearIndex, b41);
    input_t32 bVal50;
    bVal50 = b41.data[bOffset49];
    if (bVal50 >= minvalue43 && bVal50 <= maxvalue44) {
        IndexType33 bin51;
        bin51 = getBin<input_t32, IndexType33>(bVal50, minvalue43, maxvalue44, nbins42);
        atomicAdd(&smem48[bin51], getOp46(linearIndex));
    }
}
label_1:;
__syncthreads();
if (!((threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y)>=0 && (threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) < 512)) goto label_2;
for (IndexType33 i = threadIdx_x_1; i < a39.sizes[0]; i += blockDim_x_1) {
    IndexType33 aOffset52;
    aOffset52 = IndexToOffset<output_t31, IndexType33, ADims34>::get(i, a39);
    atomicAdd(&a39.data[aOffset52], smem48[i]);
}
label_2:;
}

/*
  Kernel for computing the histogram of the input.
 */
template <
    typename output_t,
    typename input_t,
    typename IndexType,
    int ADims,
    int PDims,
    int BDims,
    CUDAHistogramMemoryType MemoryType = CUDAHistogramMemoryType::MULTI_BLOCK,
    typename Op>
#ifdef __HIP_PLATFORM_HCC__
C10_LAUNCH_BOUNDS_1(512)
#endif
__global__ void kernelHistogram1D(
    TensorInfo<output_t, IndexType> a, /* output */
    TensorInfo<output_t, IndexType> p, /* partial output */
    TensorInfo<input_t, IndexType> b, /* input */
    int nbins,
    input_t minvalue,
    input_t maxvalue,
    IndexType totalElements,
    Op getOp) {
  extern __shared__ unsigned char my_smem[];
  output_t* smem = nullptr;

    ////////////////////////// Shared memory //////////////////////////
    // atomically add to block specific shared memory
    // then atomically add to the global output tensor
    smem = reinterpret_cast<output_t*>(my_smem);
    for (IndexType i = threadIdx.x; i < a.sizes[0]; i += blockDim.x) {
      smem[i] = 0;
    }
    __syncthreads();
    FOR_KERNEL_LOOP(linearIndex, totalElements) {
      // Convert `linearIndex` into an offset of `b`
      const IndexType bOffset =
          IndexToOffset<input_t, IndexType, BDims>::get(linearIndex, b);
      const input_t bVal = b.data[bOffset];
      if (bVal >= minvalue && bVal <= maxvalue) {
        // Use value at `b` as an offset of `smem`
        const IndexType bin = getBin<input_t, IndexType>(bVal, minvalue, maxvalue, nbins);
        atomicAdd(&smem[bin], getOp(linearIndex));
      }
    }
    __syncthreads();
    // NOTE: atomically update output bin count.
    //   Atomic update is imp since __syncthread() will only synchronize threads
    //   in a given block, not across blocks.
    for (IndexType i = threadIdx.x; i < a.sizes[0]; i += blockDim.x) {
      const IndexType aOffset =
          IndexToOffset<output_t, IndexType, ADims>::get(i, a);
      atomicAdd(&a.data[aOffset], smem[i]);
    }

}

inline int64_t getFreeGlobalMemory() {
  // no need to use `hipSetDevice`
  size_t free_mem, total_mem;
  hipMemGetInfo(&free_mem, &total_mem);
  AT_ASSERTM(
      hipGetLastError() == hipSuccess,
      "CUDA_tensor_histogram failed to get free global memory");
  return static_cast<int64_t>(free_mem);
}


template <typename input_hist_t>
std::tuple<Tensor, Tensor> _histc_cuda_template(
    const Tensor& self_hist,
    int64_t nbins,
    input_hist_t min,
    input_hist_t max,
    const Tensor& input,
    IntArrayRef output_size,
    bool align_corners
  ) {
  printf("2\n");
  if (nbins <= 0) {
    AT_ERROR("bins must be > 0");
  }
  Tensor output_hist = native::zeros({nbins}, device(DeviceType::CUDA).dtype(self_hist.scalar_type()));
  input_hist_t minvalue = min;
  input_hist_t maxvalue = max;
  if (min == max) {
    minvalue = *self_hist.min().cpu().data<input_hist_t>();
    maxvalue = *self_hist.max().cpu().data<input_hist_t>();
  }
  if (minvalue == maxvalue) {
    minvalue = minvalue - 1;
    maxvalue = maxvalue + 1;
  }

  printf("3\n");
  {
  checkBackend("CUDA_tensor_histogram", {output_hist, self_hist}, Backend::CUDA);
  auto totalElements = self_hist.numel();

  const dim3 block = getApplyBlock();
  dim3 grid;
  int64_t curDevice = current_device();

  grid.x = 10000;

  CUDAHistogramMemoryType memType = CUDAHistogramMemoryType::GLOBAL;
  auto maxSharedMem = getCurrentDeviceProperties()->sharedMemPerBlock;
  auto sharedMem = nbins * sizeof(input_hist_t) + 8; // 8 guard bytes
  auto maxGlobalMem = getFreeGlobalMemory();
  auto multiBlockMem = nbins * grid.x * sizeof(input_hist_t) + 8; // 8 guard bytes
  // determine memory type to use in the kernel
    printf("6\n");
  if (nbins < THRESH_NUMBER_BINS_FOR_MULTI_BLOCK_MEM &&
      sharedMem < maxSharedMem) {
    printf("shared\n");
    memType = CUDAHistogramMemoryType::SHARED;
  } else if (
      nbins < THRESH_NUMBER_BINS_FOR_GLOBAL_MEM &&
      multiBlockMem < (maxGlobalMem / 2)) {
    // check against half of free mem to be extra safe
    // due to cached allocator, we may anyway have slightly more free mem
    printf("mb\n");
    memType = CUDAHistogramMemoryType::MULTI_BLOCK;
  }

  // alloc memory for MULTI_BLOCK
  using IndexType = int64_t;
  auto aInfo = getTensorInfo<input_hist_t, IndexType>(output_hist);
  auto bInfo = getTensorInfo<input_hist_t, IndexType>(self_hist);
  TensorInfo<input_hist_t, IndexType> pInfo(nullptr, 0, {}, {});
  Tensor partial_output_hist;
  if (memType == CUDAHistogramMemoryType::MULTI_BLOCK) {
    partial_output_hist = native::zeros({grid.x, nbins}, output_hist.options());
    pInfo = getTensorInfo<input_hist_t, IndexType>(partial_output_hist);
  }

  printf("7\n");
  printf("10\n");

  Tensor output = at::empty_like(input);
  TensorArg input_arg{input, "input", 1}, output_arg{output, "output", 2};
  checkAllSameGPU("upsample_bilinear2d_out_cuda", {input_arg, output_arg});

  TORCH_CHECK(
      output_size.size() == 2,
      "It is expected output_size equals to 2, but got size ",
      output_size.size());

  int output_height = output_size[0];
  int output_width = output_size[1];

  int nbatch = input.size(0);
  int channels = input.size(1);
  int input_height = input.size(2);
  int input_width = input.size(3);

  upsample_2d_shape_check(
      input,
      Tensor(),
      nbatch,
      channels,
      input_height,
      input_width,
      output_height,
      output_width);

  output.resize_({input.size(0), input.size(1), output_height, output_width});

  AT_ASSERT(
      input_height > 0 && input_width > 0 && output_height > 0 &&
      output_width > 0);

  const int num_kernels = output_height * output_width;
  const int num_threads = std::min(
      at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock, 512);

  printf("%d %d\n", num_kernels, num_threads);
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      input.scalar_type(), "upsample_bilinear2d_out_frame", [&] {
        using accscalar_t = at::acc_type<scalar_t, true>;

        auto idata = input.packed_accessor<scalar_t, 4>();
        auto odata = output.packed_accessor<scalar_t, 4>();

        const accscalar_t rheight = area_pixel_compute_scale<accscalar_t>(
            input_height, output_height, align_corners);
        const accscalar_t rwidth = area_pixel_compute_scale<accscalar_t>(
            input_width, output_width, align_corners);

        const int num_blocks = cuda::ATenCeilDiv(num_kernels, num_threads);
        printf("%d\n", num_blocks);
        hipDeviceSynchronize();
        hipProfilerStart();
        upsample_bilinear2d_out_frame<scalar_t, accscalar_t>
            <<<num_blocks,
               num_threads,
               0,
               getStreamFromPool(true)>>>(
                num_kernels, rheight, rwidth, align_corners, idata, odata);
    static const auto getDummyOp = [] __device__(IndexType) { return 1L; };
    kernelHistogram1D<input_hist_t, input_hist_t, IndexType, 1, 2, -1, CUDAHistogramMemoryType::SHARED>
        <<<grid,
          block,
          sharedMem,
          getStreamFromPool(true)>>>(
            aInfo, pInfo, bInfo, nbins, minvalue, maxvalue, totalElements, getDummyOp);
        hipDeviceSynchronize();
      kernelHistogram1D_upsample_bilinear2d_out_frame_11<input_hist_t, input_hist_t, IndexType, 1, 2, -1, CUDAHistogramMemoryType::SHARED, decltype(getDummyOp), scalar_t, accscalar_t>
        <<<grid,
          block.x + 512,
          sharedMem,
          getStreamFromPool(true)>>>(
            aInfo, pInfo, bInfo, nbins, minvalue, maxvalue, totalElements, getDummyOp,
                num_kernels, rheight, rwidth, align_corners, idata, odata
          );

      kernelHistogram1D_upsample_bilinear2d_out_frame_0<input_hist_t, input_hist_t, IndexType, 1, 2, -1, CUDAHistogramMemoryType::SHARED, decltype(getDummyOp), scalar_t, accscalar_t>
        <<<grid,
          block.x + 512,
          sharedMem,
          getStreamFromPool(true)>>>(
            aInfo, pInfo, bInfo, nbins, minvalue, maxvalue, totalElements, getDummyOp,
                num_kernels, rheight, rwidth, align_corners, idata, odata
          );
      kernelHistogram1D_upsample_bilinear2d_out_frame_100<input_hist_t, input_hist_t, IndexType, 1, 2, -1, CUDAHistogramMemoryType::SHARED, decltype(getDummyOp), scalar_t, accscalar_t>
        <<<grid,
          512,
          sharedMem,
          getStreamFromPool(true)>>>(
            aInfo, pInfo, bInfo, nbins, minvalue, maxvalue, totalElements, getDummyOp,
                num_kernels, rheight, rwidth, align_corners, idata, odata
          );
        hipDeviceSynchronize();
        hipProfilerStop();
      });

  AT_ASSERTM(hipGetLastError() == hipSuccess, "kernelHistogram1D failed");
  return std::make_tuple(output_hist, output_hist);
}
}
} // namespace

namespace native {

std::tuple<Tensor, Tensor> _histc_upsample(
    const Tensor& self,
    int64_t nbins,
    Scalar min,
    Scalar max,
    const Tensor& input,
    IntArrayRef output_size,
    bool align_corners
  ) {
  if (self.scalar_type() == ScalarType::Half) {
    AT_ERROR("HalfTensor is not supported");
  }
    printf("0\n");
  return AT_DISPATCH_ALL_TYPES(self.scalar_type(), "histc", [&] {
    printf("1\n");
    return native::_histc_cuda_template<scalar_t>(self, nbins, min.to<scalar_t>(), max.to<scalar_t>(),
    input,
    output_size,
    align_corners
  );
  });
}

} // namespace native
} // namespace at
